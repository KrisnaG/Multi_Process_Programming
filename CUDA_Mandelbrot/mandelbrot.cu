/**
 * @file mandelbrot.cu
 * @author Krisna Gusti (kgusti@myune.edu.au)
 * @brief A parallel implementation of the Mandelbrot algorithm using Nvidia 
 * CUDA C that produces bitmap images which contain the fractal representation 
 * of the Mandelbrot set.
 * 
 * Mandelbrot algorithm taken from mandelbrot.c single thread implmentation
 * from UNE moodle.
 * 
 * Parameters:
 *      1. Width of image
 *      2. Height of image
 * 
 * Returns: 
 *      0 on Success
 * 
 * Library requirements:
 *      1. mandelbrot_util.cuh
 *      2. mandelbrot_util.cu
 *      3. bmpfile.h
 *      4. bmpfile.c
 * 
 * Build:
 *      1. make build
 * 
 * Run:
 *      1. make
 *      2. make run
 *      3. mandelbrot <width> <height>
 */

#include <stdlib.h>
#include "mandelbrot_util.cuh"

/**
 * Host main routine
 */
int main(int argc, char *argv[])
{
    int width;          /* Width of bmp image */
    int height;         /* Height of bmp image */
    bmpfile_t *bmp;     /* BMP file */
    hipError_t error;  /* Error code to check return values for CUDA calls */

    // check input arguments
    if (parse_args(argc, argv, &width, &height) < 0)
        handle_error("Usage: ./mandelbrot <width> <height>");

    long numElements = width * height;                      /* Total number of elements */
    long size = numElements * sizeof(rgb_pixel_t);          /* Size of total number of elements */
    int xoffset = -(width - 1) / 2;                         /* X offset for pixel reference */
    int yoffset = (height - 1) / 2;                         /* Y offset for pixel reference */
    bmp = bmp_create(width, height, 32);                    /* BMP object */
    rgb_pixel_t *h_pixels = (rgb_pixel_t *) malloc (size);  /* Host pixel memory */
    rgb_pixel_t *d_pixels = NULL;                           /* Device pixel memory*/
    
    // check host memory allocation
    if (h_pixels == NULL)
        handle_error("Failed to allocate host memory");

    // allocate device memory
    error = hipMalloc((void **)&d_pixels, size);
    if (error != hipSuccess) {
        free(h_pixels);
        handle_cuda_error("Failed to allocate host memory", hipGetErrorString(error));
    }
    
    // copy from host to device
    error = hipMemcpy(d_pixels, h_pixels, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        free_mem_on_error(h_pixels, d_pixels);
        handle_cuda_error("Failed to copy from host to device", hipGetErrorString(error));
    }

    // set thread and block size
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // CUDA kernel operation
    mandelbrot<<<blocksPerGrid, threadsPerBlock>>>(xoffset, yoffset, d_pixels, numElements, width);

    // check for any kernel errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        free_mem_on_error(h_pixels, d_pixels);
        handle_cuda_error("Failed to launch kernel", hipGetErrorString(error));
    }

    // copy from device to host
    error = hipMemcpy(h_pixels, d_pixels, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        free_mem_on_error(h_pixels, d_pixels);
        handle_cuda_error("Failed to copy from device to host", hipGetErrorString(error));
    }

    // set bmp pixel to construct image
    for (int col = 0; col < width; col++)
        for (int row = 0; row < height; row++)
            if (bmp_set_pixel(bmp, col, row, h_pixels[row * width + col]) == 0) {
                free_mem_on_error(h_pixels, d_pixels);
                handle_error("Failed to set pixel");
            }

    // save bmp
    if (bmp_save(bmp, FILENAME) == 0) {
        free_mem_on_error(h_pixels, d_pixels);
        handle_error("Failed to save bmp to file");
    }

    printf("Mandelbrot image created and saved in %s\n", FILENAME);

    // free bmp
    bmp_destroy(bmp);

    // free host memory
    free(h_pixels);
    
    // free device memory
    error = hipFree(d_pixels);
    if (error != hipSuccess)
        handle_cuda_error("Failed free CUDA memory", hipGetErrorString(error));

    // clean and flush  
    error = hipDeviceReset();
    if (error != hipSuccess)
        handle_cuda_error("Failed to deinitialize the device", hipGetErrorString(error));

    exit(EXIT_SUCCESS);
}