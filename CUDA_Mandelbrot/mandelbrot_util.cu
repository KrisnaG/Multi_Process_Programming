#include "hip/hip_runtime.h"
/**
 * @file mandelbrot_util.cu
 * @author Krisna Gusti (kgusti@myune.edu.au)
 * @brief CUDA mandelbrot utility implementation file.
 * 
 * Mandelbrot algorithm taken from mandelbrot.c single thread implmentation
 * from UNE moodle.
 */

#include "mandelbrot_util.cuh"

/**
 * @brief Checks input parameters are the correct, 
 * width and height.
 * 
 * @param argc number of input arguments
 * @param argv list of input arguments
 * @param width width of desired image
 * @param height height of desired image
 * @return int 0 is successful, -1 on failure
 */
int parse_args(int argc, char *argv[], int *width, int *height)
{
    if (argc != ARG_LENGTH || 
       (*width = atoi(argv[1])) <= 0 ||
       (*height = atoi(argv[2])) <= 0) {
        return FAILURE;
    }
    return SUCCESS;
}

/**
 * @brief Computes the color gradiant. Called by the device.
 * Check wiki for more details on the colour science: en.wikipedia.org/wiki/HSL_and_HSV.
 * 
 * @param color the output vector
 * @param x the gradiant (beetween 0 and 360)
 * @param min variation of the RGB channels (Move3D 0 -> 1)
 * @param max variation of the RGB channels (Move3D 0 -> 1)
 */
__device__ void GroundColorMix(double* color, double x, double min, double max)
{
  /*
   * Red = 0
   * Green = 1
   * Blue = 2
   */
    double posSlope = (max-min)/60;
    double negSlope = (min-max)/60;

    if( x < 60 )
    {
        color[0] = max;
        color[1] = posSlope*x+min;
        color[2] = min;
        return;
    }
    else if ( x < 120 )
    {
        color[0] = negSlope*x+2.0*max+min;
        color[1] = max;
        color[2] = min;
        return;
    }
    else if ( x < 180  )
    {
        color[0] = min;
        color[1] = max;
        color[2] = posSlope*x-2.0*max+min;
        return;
    }
    else if ( x < 240  )
    {
        color[0] = min;
        color[1] = negSlope*x+4.0*max+min;
        color[2] = max;
        return;
    }
    else if ( x < 300  )
    {
        color[0] = posSlope*x-4.0*max+min;
        color[1] = min;
        color[2] = max;
        return;
    }
    else
    {
        color[0] = max;
        color[1] = min;
        color[2] = negSlope*x+6*max;
        return;
    }
}

/**
 * @brief Mandelbrot kernel that calculates if a pixel (coordinate) is in
 * the mandelbrot set. 
 * 
 * @param xoffset X offset for pixel reference
 * @param yoffset Y offset for pixel reference
 * @param pixel Array of all pixels to calculate
 * @param numElements total number of pixel elements 
 * @param width width of bmp image
 */
__global__ void mandelbrot(int xoffset, int yoffset, rgb_pixel_t *pixel, long numElements, int width)
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;    /* Thread ID - pixel coordinate */
    int row = id / width;                               /* Row that the pixel is in */
    int col = id % width;                               /* Column that the pixel is in */

    if (id < numElements) {
        // Determine where in the mandelbrot set, the pixel is referencing
        double x = XCENTER + (xoffset + col) / RESOLUTION;
        double y = YCENTER + (yoffset - row) / RESOLUTION;

        // Mandelbrot stuff
        double a = 0;
        double b = 0;
        double aold = 0;
        double bold = 0;
        double zmagsqr = 0;
        int iter = 0;
        double x_col;
        double color[3];
        pixel[id] = {0, 0, 0, 0};

        // Check if the x,y coord are part of the mendelbrot set - refer to the algorithm
        while(iter < MAX_ITER && zmagsqr <= 4.0) {
            ++iter;
            a = (aold * aold) - (bold * bold) + x;
            b = 2.0 * aold * bold + y;
            zmagsqr = (a * a) + (b * b);
            aold = a;
            bold = b;	
        }

        /* Generate the colour of the pixel from the iter value */
        /* You can mess around with the colour settings to use different gradients */
        /* Colour currently maps from royal blue to red */ 
        x_col =  (COLOUR_MAX - (( ((float) iter / ((float) MAX_ITER) * GRADIENT_COLOUR_MAX))));
        GroundColorMix(color, x_col, 1, COLOUR_DEPTH);
        pixel[id].red = color[0];
        pixel[id].green = color[1];
        pixel[id].blue = color[2];
    }
}